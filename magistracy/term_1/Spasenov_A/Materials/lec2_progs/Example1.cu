#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

static void HandleError(hipError_t err,
						const char *file,
						int line) 
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( error ) (HandleError( error, __FILE__, __LINE__ ))

__global__ void addKernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

__global__ void multKernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		c[i] = a[i] * b[i];
	}
}


__global__ void func1Kernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		for (int j = 0; j < 10; j++) {
			c[i] += cosf(sqrtf(a[i]) * tanhf(b[i])) * sqrtf(j);
		}
	}
}

void initialization(float *hostA, float *hostB, const int size) {
	for (int i = 0; i < size; i++) {
		hostA[i] = sqrtf(i);
		hostB[i] = 2.*i;
	}
}

void workFunction() {
	float *hostA, *hostB, *hostC;
	float *devA, *devB, *devC;
	int arraySize = 512*50000;

	hipEvent_t GPUstart, GPUstop;
	float CPUstart, CPUstop;

	float GPUtime = 0.0f;
	float CPUtime = 0.0f;

	int N_threads = 512;
	int N_blocks;	

	size_t mem_size = sizeof(float)* arraySize;

	hostA = (float*)malloc(mem_size);
	hostB = (float*)malloc(mem_size);
	hostC = (float*)malloc(mem_size);
	
	HANDLE_ERROR(hipMalloc((void**)&devA, mem_size));
	HANDLE_ERROR(hipMalloc((void**)&devB, mem_size));
	HANDLE_ERROR(hipMalloc((void**)&devC, mem_size));

	initialization(hostA, hostB, arraySize);

	if ((arraySize % N_threads) == 0) {
		N_blocks = (arraySize / N_threads);
	}
	else {
		N_blocks = (arraySize / N_threads) + 1;
	}

	dim3 Threads(N_threads);
	dim3 Blocks(N_blocks);

	hipEventCreate(&GPUstart);
	hipEventCreate(&GPUstop);

	hipEventRecord(GPUstart, 0);

	hipMemcpy(devA, hostA, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(devB, hostB, mem_size, hipMemcpyHostToDevice);
	hipMemset(devC, 0, mem_size);


	//addKernel << < Blocks, Threads >> > (devA, devB, devC, arraySize);
	//multKernel << < Blocks, Threads >> > (devA, devB, devC, arraySize);
	func1Kernel << < Blocks, Threads >> > (devA, devB, devC, arraySize);	

	hipMemcpy(hostC, devC, mem_size, hipMemcpyDeviceToHost);	

	hipEventRecord(GPUstop, 0);
	hipEventSynchronize(GPUstop);

	hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);
	printf("GPU time : %.3f ms\n", GPUtime);

	CPUstart = clock();

	for (int i = 0; i < arraySize; i++) {
		//hostC[i] = hostA[i] + hostB[i];
		
		for (int j = 0; j < 100; j++) {
			hostC[i] += cosf(sqrtf(hostA[i]) * tanf(hostB[i])) * sqrtf(j);
		}
	}

	CPUstop = clock();
	CPUtime = 1000.*(CPUstop - CPUstart) / CLOCKS_PER_SEC;
	printf("CPU time : %.3f ms\n", CPUtime);

	printf("Rate : %.3f \n", CPUtime/GPUtime);

	free(hostA);
	free(hostB);
	free(hostC);
	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));

	HANDLE_ERROR(hipEventDestroy(GPUstart));
	HANDLE_ERROR(hipEventDestroy(GPUstop));
}

int main() {
	workFunction();
	
	return 0;
}