
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N (1024)

__global__ void mult(float *A, float *B, float *C) {
	unsigned int idx_X = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int idx_Y = threadIdx.y + blockIdx.y * blockDim.y;
	float sum = 0.;

	if ((idx_X < N) && (idx_Y < N)) {
		for (int i = 0; i < N; i++) {
			sum += A[idx_X*N + i] * B[idx_Y + i*N];
		}
		C[idx_X*N + idx_Y] = sum;
	}
}

int main(void) {
	hipEvent_t GPUstart, GPUstop;
	float GPUtime = 0.0f;

	float *hostA, *hostB;
	float *hostC;

	float *devA, *devB;
	float *devC;

	size_t mem_size = N*N*sizeof(float);

	hostA = (float *)malloc(mem_size);
	hostB = (float *)malloc(mem_size);
	hostC = (float *)malloc(mem_size);

	hipMalloc((void**)&devA, mem_size);
	hipMalloc((void**)&devB, mem_size);
	hipMalloc((void**)&devC, mem_size);

	for (int i = 0; i < N*N; i++) {
		hostA[i] = sqrtf(i);
		hostB[i] = sinf(i);
		hostC[i] = 0.;
	}

	int N_Threads = 8;
	int N_Blocks = 0;

	if (((N) % N_Threads) == 0) {
		N_Blocks = ((N) / N_Threads);
	}
	else {
		N_Blocks = ((N) / N_Threads) + 1;
	}
	dim3 Threads(N_Threads,N_Threads);
	dim3 Blocks(N_Blocks, N_Blocks);

	hipEventCreate(&GPUstart);
	hipEventCreate(&GPUstop);

	hipEventRecord(GPUstart, 0);

	hipMemcpy(devA, hostA, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(devB, hostB, mem_size, hipMemcpyHostToDevice);
	hipMemset(devC, 0, mem_size);

	mult <<< Blocks, Threads >>> (devA, devB, devC);

	hipMemcpy(hostC, devC, mem_size, hipMemcpyDeviceToHost);

	hipEventRecord(GPUstop, 0);
	hipEventSynchronize(GPUstop);

	hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);
	printf("GPU time : %.3f ms\n", GPUtime);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	free(hostA);
	free(hostB);
	free(hostC);

	return 0;
}