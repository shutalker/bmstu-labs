#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N_STREAMS 4

static void HandleError(hipError_t err,
	const char *file,
	int line)
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( error ) (HandleError( error, __FILE__, __LINE__ ))

__global__ void addKernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

__global__ void multKernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		c[i] = a[i] * b[i];
	}
}


__global__ void func1Kernel(const float *a, const float *b, float *c, const int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		for (int j = 0; j < 100; j++) {
			c[i] += cosf(sqrtf(a[i]) * tanhf(b[i])) * sqrtf(j);
		}
	}
}

void initialization(float *hostA, float *hostB, const int size) {
	for (int i = 0; i < size; i++) {
		hostA[i] = sqrtf(i);
		hostB[i] = 2.*i;
	}
}

void workFunction() {
	float *hostA, *hostB, *hostC;
	float *devA, *devB, *devC;
	int arraySize = ( 512 * 5000  ) / N_STREAMS;

	hipEvent_t GPUstart, GPUstop;
	float CPUstart, CPUstop;

	float GPUtime = 0.0f;
	float CPUtime = 0.0f;

	int N_threads = 512;
	int N_blocks;

	hipStream_t stream[N_STREAMS];

	for (int i = 0; i < N_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
	}

	size_t mem_size = sizeof(float)*arraySize;

	HANDLE_ERROR(hipHostMalloc((void**)&hostA, mem_size*N_STREAMS));
	HANDLE_ERROR(hipHostMalloc((void**)&hostB, mem_size*N_STREAMS));
	HANDLE_ERROR(hipHostMalloc((void**)&hostC, mem_size*N_STREAMS));
	
	HANDLE_ERROR(hipMalloc((void**)&devA, mem_size*N_STREAMS));
	HANDLE_ERROR(hipMalloc((void**)&devB, mem_size*N_STREAMS));
	HANDLE_ERROR(hipMalloc((void**)&devC, mem_size*N_STREAMS));
	

	initialization(hostA, hostB, arraySize);

	if ((arraySize % N_threads) == 0) {
		N_blocks = (arraySize / N_threads);
	}
	else {
		N_blocks = (arraySize / N_threads) + 1;
	}

	dim3 Threads(N_threads);
	dim3 Blocks(N_blocks);

	hipMemset(devC, 0, mem_size);

	for (int i = 0; i < N_STREAMS; ++i) {
		HANDLE_ERROR(hipMemcpyAsync(devA + i*arraySize / N_STREAMS, hostA + i*arraySize / N_STREAMS, mem_size,
			hipMemcpyHostToDevice, stream[i]) );
		HANDLE_ERROR(hipMemcpyAsync(devB + i*arraySize / N_STREAMS, hostB + i*arraySize / N_STREAMS, mem_size,
			hipMemcpyHostToDevice, stream[i]) );
	}

	hipEventCreate(&GPUstart);
	hipEventCreate(&GPUstop);

	hipEventRecord(GPUstart, 0);
	
	for (int i = 0; i < N_STREAMS; i++) {
		func1Kernel << < Blocks, Threads, 0, stream[i] >> > (devA + i*arraySize / N_STREAMS,
															 devB + i*arraySize / N_STREAMS,
															 devC + i*arraySize / N_STREAMS,
															 arraySize);
	}

	HANDLE_ERROR(hipGetLastError());

	hipEventRecord(GPUstop, 0);
	hipEventSynchronize(GPUstop);

	hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);
	printf("GPU time : %.3f ms\n", GPUtime);

	for (int i = 0; i < N_STREAMS; i++) {
		hipMemcpyAsync(hostC + i*mem_size, devC + i*mem_size, mem_size,
			hipMemcpyDeviceToHost, stream[i]);
	}

	CPUstart = clock();

	for (int i = 0; i < arraySize; i++) {
		//hostC[i] = hostA[i] + hostB[i];

		for (int j = 0; j < 100; j++) {
			hostC[i] += cosf(sqrtf(hostA[i]) * tanf(hostB[i])) * sqrtf(j);
		}
	}

	CPUstop = clock();
	CPUtime = 1000.*(CPUstop - CPUstart) / CLOCKS_PER_SEC;
	printf("CPU time : %.3f ms\n", CPUtime);

	printf("Rate : %.3f \n", CPUtime / GPUtime);
	
	HANDLE_ERROR(hipHostFree(hostA));
	HANDLE_ERROR(hipHostFree(hostB));
	HANDLE_ERROR(hipHostFree(hostC));

	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));

	HANDLE_ERROR(hipEventDestroy(GPUstart));
	HANDLE_ERROR(hipEventDestroy(GPUstop));
}

int main() {
	workFunction();

	return 0;
}