#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>
#include <string>
#include <stdio.h>

__global__ void Kernel(float *X, float *Y, float *Z) {
	unsigned int idx_X = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int idx_Y = threadIdx.y + blockIdx.y * blockDim.y;

	Z[idx_X*dim + idx_Y] = 20. + X[idx_X]*X[idx_X] +  Y[idx_Y]*Y[idx_Y] - 10.*(cosf(2.*3.14*X[idx_X]) + cosf(2.*3.14*X[idx_X]));
}

void initialization(const float leftB, const float rightB, float *X, const unsigned int dim) {
	try {
		float step = (rightB - leftB)/(float)dim;
		if (step <= 0) throw "error";

		X[0] = leftB;
		for( unsigned int i=1; i<dim ; i++) {
			X[i] = X[i-1] + step;
		}
	}
	catch (...) {
		fprintf(stderr, "step failed!");
        	exit(1);
	}
}

int main()
{
	int dim = 2048;
	size_t mem_size = sizeof(float)*dim;
	hipError_t cudaStatus;

	float *hostX, *hostY, *hostZ;
	float *devX, *devY, *devZ;

	float rightB, leftB;
	leftB = -5;
	rightB = 5;

	hostX = (float*)malloc(mem_size);
	hostY = (float*)malloc(mem_size);
	hostZ = (float*)malloc(mem_size*mem_size);

	initialization(leftB, rightB, hostX, dim);

	//memcpy(hostY,hostX,mem_size);

	hipMalloc((void**)&devX, mem_size);
	hipMalloc((void**)&devY, mem_size);
	hipMalloc((void**)&devZ, mem_size*mem_size);

	hipMemcpy(devX, hostX, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(devY, devX, mem_size, hipMemcpyDeviceToDevice);

	dim3 N_Grid  (dim/32,dim/32,1);
	dim3 N_Block (48,48,1);
	
	Kernel <<< N_Grid, N_Block >>> (devX,devY,devZ);
	cudaStatus = hipGetLastError();

	if(cudaStatus != hipSuccess) {
		printf("Last error: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}

	hipMemcpy(hostZ, devZ, mem_size*mem_size, hipMemcpyDeviceToHost);

	for(unsigned int i=0; i<dim*dim; i++) {
		std::cout << "i: " << hostZ[i] << std::endl;
	}

	hipFree(devX);
    hipFree(devY);
    hipFree(devZ);
	free(hostX);
	free(hostY);
	free(hostZ);

    return 0;
}
